#include "hip/hip_runtime.h"
#ifndef _MATMUL_KERNEL_H_
#define _MATMUL_KERNEL_H_

#include <stdio.h>
#include "matrix.h"
#include "math.h"

////////////////////////UTILITY FUNCTIONS////////////////////////

__device__ float mat_get(Matrix& m, int y, int x)
{
    return m.elements[y * m.width + x];
}

__device__ void mat_set(Matrix& m, int y, int x, float value)
{
    m.elements[y * m.width + x] = value;
}

////////////////////GPU-BASED IMPLEMENTATION//////////////////////

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc((void**)&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    // Load B to device memory
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc((void**)&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc((void**)&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(min(B.width, 16), min(A.height, 16));
    dim3 dimGrid((int)ceil(1.0 * B.width / dimBlock.x), (int)ceil(1.0 * A.height / dimBlock.y));
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
}

// Matrix multiplication kernel called by MatrixMul()
// also see comments to Playground.Conflux\SampleKernels\MatMulKernel.cs
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (A.height <= row || B.width <= col) return;
    
    float Cvalue = 0;
    for (int dim = 0; dim < A.width; ++dim)
    {
        Cvalue += mat_get(A, row, dim) * mat_get(B, dim, col);
    }
    
    mat_set(C, row, col, Cvalue);
}

#endif