#include "hip/hip_runtime.h"
#ifndef _MATMUL_KERNEL_FAST_H_
#define _MATMUL_KERNEL_FAST_H_

#include <stdio.h>
#include "matrix.h"

////////////////////GPU-BASED IMPLEMENTATION//////////////////////

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel_Fast(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul_Fast(const Matrix A, const Matrix B, Matrix C)
{
    // Load A to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc((void**)&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    // Load B to device memory
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc((void**)&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc((void**)&d_C.elements, size);

    // Invoke kernel
    int mindim = min(min(B.width, 16), min(A.height, 16));
    dim3 dimBlock(mindim, mindim);
    dim3 dimGrid((int)ceil(1.0 * B.width / dimBlock.x), (int)ceil(1.0 * A.height / dimBlock.y));
    MatMulKernel_Fast<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
}

////////////////////FRAGMENT OF THE MATRIX////////////////////////

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int top;
    int left;
    int height;
    int width;
    int stride;
    float* elements;
} SubMatrix;

__device__ SubMatrix sub_init(Matrix m, int blockRow, int blockCol)
{
    SubMatrix msub;
    msub.top = blockRow * blockDim.y;
    msub.left = blockCol * blockDim.x;
    msub.height = min(blockDim.y, m.height - msub.top);
    msub.width = min(blockDim.x, m.width - msub.left);
    msub.stride = m.width;
    msub.elements = m.elements;
    return msub;
}

__device__ float sub_get(const SubMatrix msub, int row, int col)
{
    return msub.elements[(msub.top + row) * msub.stride + (msub.left + col)];
}

__device__ void sub_set(SubMatrix msub, int row, int col, float value)
{
    msub.elements[(msub.top + row) * msub.stride + (msub.left + col)] = value;
}

////////////////////GPU-BASED IMPLEMENTATION//////////////////////

// Matrix multiplication kernel called by MatMul_Fast()
// also see comments to Playground.Conflux\SampleKernels\MatMulKernel_Fast.cs
__global__ void MatMulKernel_Fast(Matrix A, Matrix B, Matrix C)
{
    int blockSize = blockDim.x;
    
    float c_value = 0;
    for (int i = 0; i < (int)ceil(1.0 * A.width / blockSize); ++i) 
    {
        __shared__ float asub_shared[16][16];
        __shared__ float bsub_shared[16][16];

        SubMatrix asub = sub_init(A, blockIdx.y, i);
        if (asub.height > threadIdx.y && asub.width > threadIdx.x)
            asub_shared[threadIdx.y][threadIdx.x] = sub_get(asub, threadIdx.y, threadIdx.x);

        SubMatrix bsub = sub_init(B, i, blockIdx.x);
        if (bsub.height > threadIdx.y && bsub.width > threadIdx.x)
            bsub_shared[threadIdx.y][threadIdx.x] = sub_get(bsub, threadIdx.y, threadIdx.x);

        __syncthreads();

        int stripLen = min(A.width - i * blockSize, blockSize);
        for (int j = 0; j < stripLen; ++j)
            c_value += asub_shared[threadIdx.y][j] * bsub_shared[j][threadIdx.x];

        __syncthreads();
    }
    
    SubMatrix csub = sub_init(C, blockIdx.y, blockIdx.x);
    if (csub.height > threadIdx.y && csub.width > threadIdx.x)
        sub_set(csub, threadIdx.y, threadIdx.x, c_value);
}

#endif